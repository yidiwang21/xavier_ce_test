#include "include.cuh"

extern const char *_cublasGetStatusString(hipblasStatus_t status);

void GTC_Gemm (const float *A, const float *B, float *C, const int M, const int K, const int N) {
    int lda = M, ldb = K, ldc = M;
    const float alpha = 1.0;
    const float beta = 0.0;

    // create CUBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // set math mode to enable tensor cores
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH) ;

    hipblasStatus_t status = hipblasSgemm(handle, 
                    HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    M, N, K, 
                    &alpha, 
                    A, lda, 
                    B, ldb, 
                    &beta, 
                    C, ldc);

    if (status != HIPBLAS_STATUS_SUCCESS) 
        printf("CUDA Error: %d\n", _cublasGetStatusString(status));

    // do not destroy the handle if more than one mmul need to be done
    hipblasDestroy(handle);
}