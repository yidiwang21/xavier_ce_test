#include "include.cuh"
#include "ce_func.cuh"
#include "support.cuh"
#include "sm_alloc.cuh"
#include "common_snippet.cuh"
#include "hip/hip_runtime_api.h"
#include <thread>

int gflag = 0;
std::mutex mtx;

int main(int argc, char *argv[]) {
    int opt;
    enum exec_mode EM = INDIVIDUAL;
    int loop_num = 1;
    enum computing_elem CE = GPU_REG_CORES;
    while ((opt = getopt (argc, argv, "e:l: ")) != EOF) {
        switch (opt) {
            case 'e':
                if (atoi(optarg) == 0) CE = CPU_CORES; 
                else if (atoi(optarg) == 1) CE = GPU_REG_CORES;
                else if (atoi(optarg) == 2) CE = GPU_TENSOR_CORES;
                else if (atoi(optarg) == -1) CE = TEST_OUTPUT;
                break;
            case 'l':
                loop_num = atoi(optarg);
                break;
            case '?':
                fprintf(stderr, "# Usage: \n -e <computing_elem> -l <loop number>");
            default:
                printf("\n"); abort();
        }
    }

    int sm = 8;
    std::string outfile;
    switch (CE) {
        case GPU_REG_CORES:
            printf("Computing element: GPU regular cores\n"); fflush(stdout);
            outfile = "power_gpu_reg_SM_" + std::to_string(sm) + ".txt";
            break;
        case GPU_TENSOR_CORES:
            printf("Computing element: GPU tensor cores\n"); fflush(stdout);
            outfile = "power_gpu_tensor_SM_" + std::to_string(sm) + ".txt";
            break;
    }

    hipError_t cuda_ret;

    float time1, time2;
    Timer timer;
    startTime(&timer);

    MM_INIT_PARAM();
    MM_CREATE_CUDA_STREAM();
    MM_CREATE_HANDLE();
    MM_ALLOC_HOST_VARS();
    MM_ALLOC_DEVICE_VARS();
    MM_COPY_MEMORY_FROM_HOST_TO_DEVICE();

    hipDeviceSynchronize();

    int fd = open("/sys/bus/i2c/drivers/ina3221x/1-0040/iio:device0/in_power0_input", O_RDONLY | O_NONBLOCK);
    printf("Creating thread for power reading...");
    std::thread power_thread(get_data_from_sensor, fd, outfile, 0);
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("##################################################\n");
    
    stopTime(&timer);
    time1 = elapsedTime(timer);
    printf("Run GEMM for %d times...", loop_num);
    if (CE == GPU_REG_CORES) {
        for (int i = 0; i < loop_num; i++) { 
            GRC_GEMM();
            hipDeviceSynchronize();
        }
    }else if (CE == GPU_TENSOR_CORES) {
        for (int i = 0; i < loop_num; i++) {
            GTC_GEMM();
            hipDeviceSynchronize();
        }
    }
    
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    time2 = elapsedTime(timer);


    // printf("Launching dummy_kernel..."); fflush(stdout);
    // dummy_kernel <<< MAX_SM * reg_num_block_per_sm, 1, 0, dummy_stream >>> ();
    // if (EM == INDIVIDUAL) {
    //     cuda_ret = hipDeviceSynchronize();
    //     stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    //     if (cuda_ret != hipSuccess) {
    //         printf("Failed to launch dummy_kernel.\n");
    //         exit(1);
    //     }
    // }
    // printf("Launching GEMM on tensor cores..."); fflush(stdout);
    // stopTime(&timer);
    // time1 = elapsedTime(timer);
    // GTC_GEMM();

    // hipStreamSynchronize(mm_stream);
    // hipStreamSynchronize(mm_stream);

    mtx.lock();
    gflag = 1;
    mtx.unlock();
    power_thread.join();

    std::ofstream out;
    out.open(outfile, std::ios::app);
    out << "time1: " << 0 << std::endl;
    out << "time2: " << time2 << std::endl;


    MM_COPY_MEMORY_FROM_DEVICE_TO_HOST();
    hipDeviceSynchronize();

    MM_FREE_DEVICE_VARS();
    MM_FREE_HOST_VARS();

    MM_DESTROY_HANDLE();

    hipDeviceSynchronize();

    
    hipProfilerStop();
    return 0;

}